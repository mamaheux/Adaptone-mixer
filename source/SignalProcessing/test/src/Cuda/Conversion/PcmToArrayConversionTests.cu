#include "hip/hip_runtime.h"
#include <SignalProcessing/Cuda/Conversion/PcmToArrayConversion.h>

#include <gtest/gtest.h>

using namespace adaptone;
using namespace std;

static constexpr double MaxAbsError = 0.01;

template<class T>
__global__ void convert(const uint8_t* inputBytes, T* output, std::size_t frameSampleCount, std::size_t channelCount,
    PcmToArrayConversionFunctionPointer<T> conversionFunction)
{
    conversionFunction(inputBytes, output, frameSampleCount, channelCount);
}

TEST(PcmToArrayConversionTests, convertSigned8_shouldConvertTheDataToFloatingPointArray)
{
    PcmToArrayConversionFunctionPointer<float> conversionFunction =
        getPcmToArrayConversionFunctionPointer<float>(PcmAudioFrame::Format::Signed8);

    std::size_t frameSampleCount = 3;
    std::size_t channelCount = 2;
    int8_t* input;
    float* output;

    hipMallocManaged(reinterpret_cast<void**>(&input), frameSampleCount * channelCount * sizeof(int8_t));
    hipMallocManaged(reinterpret_cast<void**>(&output), frameSampleCount * channelCount * sizeof(float));

    input[0] = -128;
    input[1] = 0;
    input[2] = 127;
    input[3] = 64;
    input[4] = -64;
    input[5] = 32;

    convert<<<1, 256>>>(reinterpret_cast<uint8_t*>(input), output, frameSampleCount, channelCount, conversionFunction);
    hipDeviceSynchronize();

    EXPECT_NEAR(output[0], -1, MaxAbsError);
    EXPECT_NEAR(output[1], 1, MaxAbsError);
    EXPECT_NEAR(output[2], -0.5, MaxAbsError);

    EXPECT_NEAR(output[3], 0, MaxAbsError);
    EXPECT_NEAR(output[4], 0.5, MaxAbsError);
    EXPECT_NEAR(output[5], 0.25, MaxAbsError);


    hipFree(input);
    hipFree(output);
}

TEST(PcmToArrayConversionTests, convertSigned16_shouldConvertTheDataToFloatingPointArray)
{
    PcmToArrayConversionFunctionPointer<float> conversionFunction =
        getPcmToArrayConversionFunctionPointer<float>(PcmAudioFrame::Format::Signed16);

    std::size_t frameSampleCount = 3;
    std::size_t channelCount = 2;
    int16_t* input;
    float* output;

    hipMallocManaged(reinterpret_cast<void**>(&input), frameSampleCount * channelCount * sizeof(int16_t));
    hipMallocManaged(reinterpret_cast<void**>(&output), frameSampleCount * channelCount * sizeof(float));

    input[0] = -32768;
    input[1] = 0;
    input[2] = 32767;
    input[3] = 16384;
    input[4] = -16384;
    input[5] = 8192;

    convert<<<1, 256>>>(reinterpret_cast<uint8_t*>(input), output, frameSampleCount, channelCount, conversionFunction);
    hipDeviceSynchronize();

    EXPECT_NEAR(output[0], -1, MaxAbsError);
    EXPECT_NEAR(output[1], 1, MaxAbsError);
    EXPECT_NEAR(output[2], -0.5, MaxAbsError);

    EXPECT_NEAR(output[3], 0, MaxAbsError);
    EXPECT_NEAR(output[4], 0.5, MaxAbsError);
    EXPECT_NEAR(output[5], 0.25, MaxAbsError);

    hipFree(input);
    hipFree(output);
}

TEST(PcmToArrayConversionTests, convertSigned24_shouldConvertTheDataToFloatingPointArray)
{
    PcmToArrayConversionFunctionPointer<float> conversionFunction =
        getPcmToArrayConversionFunctionPointer<float>(PcmAudioFrame::Format::Signed24);

    std::size_t frameSampleCount = 3;
    std::size_t channelCount = 2;
    uint8_t* input;
    float* output;

    hipMallocManaged(reinterpret_cast<void**>(&input), frameSampleCount * channelCount * 3);
    hipMallocManaged(reinterpret_cast<void**>(&output), frameSampleCount * channelCount * sizeof(float));

    //-8388608
    input[0] = 0x00;
    input[1] = 0x00;
    input[2] = 0x80;

    //0
    input[3] = 0x00;
    input[4] = 0x00;
    input[5] = 0x00;

    //8388607
    input[6] = 0xff;
    input[7] = 0xff;
    input[8] = 0x7f;

    //4194304
    input[9] = 0x00;
    input[10] = 0x00;
    input[11] = 0x40;

    //-4194304
    input[12] = 0x00;
    input[13] = 0x00;
    input[14] = 0xC0;

    //2097152
    input[15] = 0x00;
    input[16] = 0x00;
    input[17] = 0x20;

    convert<<<1, 256>>>(input, output, frameSampleCount, channelCount, conversionFunction);
    hipDeviceSynchronize();

    EXPECT_NEAR(output[0], -1, MaxAbsError);
    EXPECT_NEAR(output[1], 1, MaxAbsError);
    EXPECT_NEAR(output[2], -0.5, MaxAbsError);

    EXPECT_NEAR(output[3], 0, MaxAbsError);
    EXPECT_NEAR(output[4], 0.5, MaxAbsError);
    EXPECT_NEAR(output[5], 0.25, MaxAbsError);

    hipFree(input);
    hipFree(output);
}

TEST(PcmToArrayConversionTests, convertSignedPadded24_shouldConvertTheDataToFloatingPointArray)
{
    PcmToArrayConversionFunctionPointer<float> conversionFunction =
        getPcmToArrayConversionFunctionPointer<float>(PcmAudioFrame::Format::SignedPadded24);

    std::size_t frameSampleCount = 3;
    std::size_t channelCount = 2;
    int32_t* input;
    float* output;

    hipMallocManaged(reinterpret_cast<void**>(&input), frameSampleCount * channelCount * sizeof(int32_t));
    hipMallocManaged(reinterpret_cast<void**>(&output), frameSampleCount * channelCount * sizeof(float));

    input[0] = -8388608;
    input[1] = 0;
    input[2] = 8388607;
    input[3] = 4194304;
    input[4] = -4194304;
    input[5] = 2097152;

    convert<<<1, 256>>>(reinterpret_cast<uint8_t*>(input), output, frameSampleCount, channelCount, conversionFunction);
    hipDeviceSynchronize();

    EXPECT_NEAR(output[0], -1, MaxAbsError);
    EXPECT_NEAR(output[1], 1, MaxAbsError);
    EXPECT_NEAR(output[2], -0.5, MaxAbsError);

    EXPECT_NEAR(output[3], 0, MaxAbsError);
    EXPECT_NEAR(output[4], 0.5, MaxAbsError);
    EXPECT_NEAR(output[5], 0.25, MaxAbsError);

    hipFree(input);
    hipFree(output);
}

TEST(PcmToArrayConversionTests, convertSigned32_shouldConvertTheDataToFloatingPointArray)
{
    PcmToArrayConversionFunctionPointer<float> conversionFunction =
        getPcmToArrayConversionFunctionPointer<float>(PcmAudioFrame::Format::Signed32);

    std::size_t frameSampleCount = 3;
    std::size_t channelCount = 2;
    int32_t* input;
    float* output;

    hipMallocManaged(reinterpret_cast<void**>(&input), frameSampleCount * channelCount * sizeof(int32_t));
    hipMallocManaged(reinterpret_cast<void**>(&output), frameSampleCount * channelCount * sizeof(float));

    input[0] = -2147483648;
    input[1] = 0;
    input[2] = 2147483647;
    input[3] = 1073741824;
    input[4] = -1073741824;
    input[5] = 536870912;

    convert<<<1, 256>>>(reinterpret_cast<uint8_t*>(input), output, frameSampleCount, channelCount, conversionFunction);
    hipDeviceSynchronize();

    EXPECT_NEAR(output[0], -1, MaxAbsError);
    EXPECT_NEAR(output[1], 1, MaxAbsError);
    EXPECT_NEAR(output[2], -0.5, MaxAbsError);

    EXPECT_NEAR(output[3], 0, MaxAbsError);
    EXPECT_NEAR(output[4], 0.5, MaxAbsError);
    EXPECT_NEAR(output[5], 0.25, MaxAbsError);

    hipFree(input);
    hipFree(output);
}

TEST(PcmToArrayConversionTests, convertUnsigned8_shouldConvertTheDataToFloatingPointArray)
{
    PcmToArrayConversionFunctionPointer<float> conversionFunction =
        getPcmToArrayConversionFunctionPointer<float>(PcmAudioFrame::Format::Unsigned8);

    std::size_t frameSampleCount = 3;
    std::size_t channelCount = 2;
    uint8_t* input;
    float* output;

    hipMallocManaged(reinterpret_cast<void**>(&input), frameSampleCount * channelCount * sizeof(uint8_t));
    hipMallocManaged(reinterpret_cast<void**>(&output), frameSampleCount * channelCount * sizeof(float));

    input[0] = 0;
    input[1] = 128;
    input[2] = 255;
    input[3] = 192;
    input[4] = 64;
    input[5] = 160;

    convert<<<1, 256>>>(input, output, frameSampleCount, channelCount, conversionFunction);
    hipDeviceSynchronize();

    EXPECT_NEAR(output[0], -1, MaxAbsError);
    EXPECT_NEAR(output[1], 1, MaxAbsError);
    EXPECT_NEAR(output[2], -0.5, MaxAbsError);

    EXPECT_NEAR(output[3], 0, MaxAbsError);
    EXPECT_NEAR(output[4], 0.5, MaxAbsError);
    EXPECT_NEAR(output[5], 0.25, MaxAbsError);

    hipFree(input);
    hipFree(output);
}

TEST(PcmToArrayConversionTests, convertUnsigned16_shouldConvertTheDataToFloatingPointArray)
{
    PcmToArrayConversionFunctionPointer<float> conversionFunction =
        getPcmToArrayConversionFunctionPointer<float>(PcmAudioFrame::Format::Unsigned16);

    std::size_t frameSampleCount = 3;
    std::size_t channelCount = 2;
    uint16_t* input;
    float* output;

    hipMallocManaged(reinterpret_cast<void**>(&input), frameSampleCount * channelCount * sizeof(uint16_t));
    hipMallocManaged(reinterpret_cast<void**>(&output), frameSampleCount * channelCount * sizeof(float));

    input[0] = 0;
    input[1] = 32768;
    input[2] = 65535;
    input[3] = 49152;
    input[4] = 16384;
    input[5] = 40960;

    convert<<<1, 256>>>(reinterpret_cast<uint8_t*>(input), output, frameSampleCount, channelCount, conversionFunction);
    hipDeviceSynchronize();

    EXPECT_NEAR(output[0], -1, MaxAbsError);
    EXPECT_NEAR(output[1], 1, MaxAbsError);
    EXPECT_NEAR(output[2], -0.5, MaxAbsError);

    EXPECT_NEAR(output[3], 0, MaxAbsError);
    EXPECT_NEAR(output[4], 0.5, MaxAbsError);
    EXPECT_NEAR(output[5], 0.25, MaxAbsError);

    hipFree(input);
    hipFree(output);
}

TEST(PcmToArrayConversionTests, convertUnsigned24_shouldConvertTheDataToFloatingPointArray)
{
    PcmToArrayConversionFunctionPointer<float> conversionFunction =
        getPcmToArrayConversionFunctionPointer<float>(PcmAudioFrame::Format::Unsigned24);

    std::size_t frameSampleCount = 3;
    std::size_t channelCount = 2;
    uint8_t* input;
    float* output;

    hipMallocManaged(reinterpret_cast<void**>(&input), frameSampleCount * channelCount * 3);
    hipMallocManaged(reinterpret_cast<void**>(&output), frameSampleCount * channelCount * sizeof(float));

    //0
    input[0] = 0x00;
    input[1] = 0x00;
    input[2] = 0x00;

    //8388608
    input[3] = 0x00;
    input[4] = 0x00;
    input[5] = 0x80;

    //16777215
    input[6] = 0xff;
    input[7] = 0xff;
    input[8] = 0xff;

    //12582912
    input[9] = 0x00;
    input[10] = 0x00;
    input[11] = 0xc0;

    //4194304
    input[12] = 0x00;
    input[13] = 0x00;
    input[14] = 0x40;

    //10485760
    input[15] = 0x00;
    input[16] = 0x00;
    input[17] = 0xa0;

    convert<<<1, 256>>>(input, output, frameSampleCount, channelCount, conversionFunction);
    hipDeviceSynchronize();

    EXPECT_NEAR(output[0], -1, MaxAbsError);
    EXPECT_NEAR(output[1], 1, MaxAbsError);
    EXPECT_NEAR(output[2], -0.5, MaxAbsError);

    EXPECT_NEAR(output[3], 0, MaxAbsError);
    EXPECT_NEAR(output[4], 0.5, MaxAbsError);
    EXPECT_NEAR(output[5], 0.25, MaxAbsError);

    hipFree(input);
    hipFree(output);
}

TEST(PcmToArrayConversionTests, convertUnsignedPadded24_shouldConvertTheDataToFloatingPointArray)
{
    PcmToArrayConversionFunctionPointer<float> conversionFunction =
        getPcmToArrayConversionFunctionPointer<float>(PcmAudioFrame::Format::UnsignedPadded24);

    std::size_t frameSampleCount = 3;
    std::size_t channelCount = 2;
    uint32_t* input;
    float* output;

    hipMallocManaged(reinterpret_cast<void**>(&input), frameSampleCount * channelCount * sizeof(uint32_t));
    hipMallocManaged(reinterpret_cast<void**>(&output), frameSampleCount * channelCount * sizeof(float));

    input[0] = 0;
    input[1] = 8388608;
    input[2] = 16777215;
    input[3] = 12582912;
    input[4] = 4194304;
    input[5] = 10485760;

    convert<<<1, 256>>>(reinterpret_cast<uint8_t*>(input), output, frameSampleCount, channelCount, conversionFunction);
    hipDeviceSynchronize();

    EXPECT_NEAR(output[0], -1, MaxAbsError);
    EXPECT_NEAR(output[1], 1, MaxAbsError);
    EXPECT_NEAR(output[2], -0.5, MaxAbsError);

    EXPECT_NEAR(output[3], 0, MaxAbsError);
    EXPECT_NEAR(output[4], 0.5, MaxAbsError);
    EXPECT_NEAR(output[5], 0.25, MaxAbsError);

    hipFree(input);
    hipFree(output);
}

TEST(PcmToArrayConversionTests, convertUnsigned32_shouldConvertTheDataToFloatingPointArray)
{
    PcmToArrayConversionFunctionPointer<float> conversionFunction =
        getPcmToArrayConversionFunctionPointer<float>(PcmAudioFrame::Format::Unsigned32);

    std::size_t frameSampleCount = 3;
    std::size_t channelCount = 2;
    uint32_t* input;
    float* output;

    hipMallocManaged(reinterpret_cast<void**>(&input), frameSampleCount * channelCount * sizeof(uint32_t));
    hipMallocManaged(reinterpret_cast<void**>(&output), frameSampleCount * channelCount * sizeof(float));

    input[0] = 0;
    input[1] = 2147483648;
    input[2] = 4294967295;
    input[3] = 3221225472;
    input[4] = 1073741824;
    input[5] = 2684354560;

    convert<<<1, 256>>>(reinterpret_cast<uint8_t*>(input), output, frameSampleCount, channelCount, conversionFunction);
    hipDeviceSynchronize();

    EXPECT_NEAR(output[0], -1, MaxAbsError);
    EXPECT_NEAR(output[1], 1, MaxAbsError);
    EXPECT_NEAR(output[2], -0.5, MaxAbsError);

    EXPECT_NEAR(output[3], 0, MaxAbsError);
    EXPECT_NEAR(output[4], 0.5, MaxAbsError);
    EXPECT_NEAR(output[5], 0.25, MaxAbsError);

    hipFree(input);
    hipFree(output);
}

TEST(PcmToArrayConversionTests, convertFloat_shouldConvertTheDataToFloatingPointArray)
{
    PcmToArrayConversionFunctionPointer<float> conversionFunction =
        getPcmToArrayConversionFunctionPointer<float>(PcmAudioFrame::Format::Float);

    std::size_t frameSampleCount = 3;
    std::size_t channelCount = 2;
    float* input;
    float* output;

    hipMallocManaged(reinterpret_cast<void**>(&input), frameSampleCount * channelCount * sizeof(float));
    hipMallocManaged(reinterpret_cast<void**>(&output), frameSampleCount * channelCount * sizeof(float));

    input[0] = -1;
    input[1] = 0;
    input[2] = 1;
    input[3] = 0.5;
    input[4] = -0.5;
    input[5] = 0.25;

    convert<<<1, 256>>>(reinterpret_cast<uint8_t*>(input), output, frameSampleCount, channelCount, conversionFunction);
    hipDeviceSynchronize();

    EXPECT_NEAR(output[0], -1, MaxAbsError);
    EXPECT_NEAR(output[1], 1, MaxAbsError);
    EXPECT_NEAR(output[2], -0.5, MaxAbsError);

    EXPECT_NEAR(output[3], 0, MaxAbsError);
    EXPECT_NEAR(output[4], 0.5, MaxAbsError);
    EXPECT_NEAR(output[5], 0.25, MaxAbsError);

    hipFree(input);
    hipFree(output);
}

TEST(PcmToArrayConversionTests, convertDouble_shouldConvertTheDataToFloatingPointArray)
{
    PcmToArrayConversionFunctionPointer<float> conversionFunction =
        getPcmToArrayConversionFunctionPointer<float>(PcmAudioFrame::Format::Double);

    std::size_t frameSampleCount = 3;
    std::size_t channelCount = 2;
    double* input;
    float* output;

    hipMallocManaged(reinterpret_cast<void**>(&input), frameSampleCount * channelCount * sizeof(double));
    hipMallocManaged(reinterpret_cast<void**>(&output), frameSampleCount * channelCount * sizeof(float));

    input[0] = -1;
    input[1] = 0;
    input[2] = 1;
    input[3] = 0.5;
    input[4] = -0.5;
    input[5] = 0.25;

    convert<<<1, 256>>>(reinterpret_cast<uint8_t*>(input), output, frameSampleCount, channelCount, conversionFunction);
    hipDeviceSynchronize();

    EXPECT_NEAR(output[0], -1, MaxAbsError);
    EXPECT_NEAR(output[1], 1, MaxAbsError);
    EXPECT_NEAR(output[2], -0.5, MaxAbsError);

    EXPECT_NEAR(output[3], 0, MaxAbsError);
    EXPECT_NEAR(output[4], 0.5, MaxAbsError);
    EXPECT_NEAR(output[5], 0.25, MaxAbsError);

    hipFree(input);
    hipFree(output);
}
