#include "hip/hip_runtime.h"
#include <SignalProcessing/Cuda/Processing/MixProcessing.h>

#include <gtest/gtest.h>

using namespace adaptone;
using namespace std;

template<class T>
__global__ void processMixKernel(T* inputFrame, T* outputFrame, T* gains, std::size_t frameSampleCount,
    std::size_t inputChannelCount, std::size_t outputChannelCount)
{
    processMix(inputFrame, outputFrame, gains, frameSampleCount, inputChannelCount, outputChannelCount);
}

TEST(MixProcessingTests, processMix_shouldMixTheInput)
{
    size_t frameSampleCount = 4;
    size_t inputChannelCount = 3;
    size_t outputChannelCount = 2;
    float* inputFrame;
    float* outputFrame;
    float* gains;

    hipMallocManaged(reinterpret_cast<void**>(&inputFrame), frameSampleCount * inputChannelCount * sizeof(float));
    hipMallocManaged(reinterpret_cast<void**>(&outputFrame), frameSampleCount * outputChannelCount * sizeof(float));
    hipMallocManaged(reinterpret_cast<void**>(&gains), inputChannelCount * outputChannelCount * sizeof(float));

    inputFrame[0] = -128;
    inputFrame[1] = 1;
    inputFrame[2] = 127;
    inputFrame[3] = 16;

    inputFrame[4] = 64;
    inputFrame[5] = -64;
    inputFrame[6] = 32;
    inputFrame[7] = 32;

    inputFrame[8] = -32;
    inputFrame[9] = 127;
    inputFrame[10] = 64;
    inputFrame[11] = 64;

    gains[0] = 0.5;
    gains[1] = 2;
    gains[2] = -0.5;

    gains[3] = 0.25;
    gains[4] = 1.5;
    gains[5] = -1.5;

    processMixKernel<<<1, 256>>>(inputFrame, outputFrame, gains, frameSampleCount, inputChannelCount, outputChannelCount);
    hipDeviceSynchronize();

    EXPECT_EQ(outputFrame[0], 80);
    EXPECT_EQ(outputFrame[1], -191);
    EXPECT_EQ(outputFrame[2], 95.5);
    EXPECT_EQ(outputFrame[3], 40);

    EXPECT_EQ(outputFrame[4], 112);
    EXPECT_EQ(outputFrame[5], -286.25);
    EXPECT_EQ(outputFrame[6], -16.25);
    EXPECT_EQ(outputFrame[7], -44);

    hipFree(inputFrame);
    hipFree(outputFrame);
    hipFree(gains);
}